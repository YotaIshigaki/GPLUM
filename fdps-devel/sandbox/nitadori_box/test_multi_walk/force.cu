#include "hip/hip_runtime.h"

#include "class.hpp"
#include "force.hpp"

#define N_THREAD_GPU 64
const int N_WALK_LIMIT = 1000;
const int NI_LIMIT = N_WALK_LIMIT*1000;
const int NJ_LIMIT = N_WALK_LIMIT*10000;

#if 0
#  include <cutil.h>
#else
#  include <hip/hip_runtime_api.h>
#  define CUDA_SAFE_CALL checkCudaErrors
#endif

class EpiGPU{
public:
    float2 pos[3];
    int id_walk;
};

class EpjGPU{
public:
    float mass;
    float2 pos[3];
};

class ForceGPU{
public:
    float2 acc[3];
    float2 pot;
};

static float2 float2_split(double x){
    const int shift = 20;
    float2 ret;
    x *= (1<<shift);
    double xi = (int)x;
    double xf = x - xi;
    ret.x = xi * (1./(1<<shift));
    ret.y = xf * (1./(1<<shift));
    return ret;
}

__device__ float2 float2_accum(float2 acc, float x){
    float tmp = acc.x + x;
    acc.y -= (tmp - acc.x) - x;
    acc.x = tmp;
    return acc;
}

__device__ float2 float2_regularize(float2 acc){
    float tmp = acc.x + acc.y;
    acc.y = acc.y -(tmp - acc.x);
    acc.x = tmp;
    return acc;
}

__global__ void ForceKernel(const EpiGPU * epi,
                            const int    * ni_disp,
                            const EpjGPU * epj, 
                            const int    * nj_disp,
                            ForceGPU     * force,
                            const float eps2){
    int id_i = blockDim.x * blockIdx.x + threadIdx.x;
    const EpiGPU & ip = epi[id_i];
    float2 poti;
    float2 acci[3];
    poti = acci[0] = acci[1] = acci[2] = make_float2(0.0, 0.0);
    const int j_head = nj_disp[ip.id_walk];
    const int j_tail = nj_disp[ip.id_walk+1];
    const int nj = j_tail - j_head;
    for(int j=j_head; j<j_tail; j++){
        //int j = j_head + threadIdx.x%nj;
        //for(int jtmp=0; jtmp<nj; jtmp++){
        EpjGPU jp = epj[j];
        const float dx = (jp.pos[0].x - ip.pos[0].x) + (jp.pos[0].y - ip.pos[0].y);
        const float dy = (jp.pos[1].x - ip.pos[1].x) + (jp.pos[1].y - ip.pos[1].y);
        const float dz = (jp.pos[2].x - ip.pos[2].x) + (jp.pos[2].y - ip.pos[2].y); // 9op
        const float r2 = ((eps2 + dx*dx) + dy*dy) + dz*dz; // 15op
        const float r_inv = rsqrtf(r2); // 15op + 1rsqrt
        const float pij = jp.mass * r_inv * (r2 > eps2); 
        const float r2_inv = r_inv * r_inv; 
        const float pij_r3_inv = pij * r2_inv; 
        const float ax = pij_r3_inv * dx; 
        const float ay = pij_r3_inv * dy;
        const float az = pij_r3_inv * dz; //21op + 1rsqrt
        poti = float2_accum(poti, pij);
        acci[0] = float2_accum(acci[0], ax);
        acci[1] = float2_accum(acci[1], ay);
        acci[2] = float2_accum(acci[2], az); // 33op + 1rsqrt
        //j = (j+1)>=j_tail ? j_head : j+1;
    }
    poti = float2_regularize(poti);
    acci[0] = float2_regularize(acci[0]);
    acci[1] = float2_regularize(acci[1]);
    acci[2] = float2_regularize(acci[2]);
    force[id_i].pot = poti;
    force[id_i].acc[0] = acci[0];
    force[id_i].acc[1] = acci[1];
    force[id_i].acc[2] = acci[2];
}

static ForceGPU * force_d;
static ForceGPU * force_h;
static EpiGPU * epi_d;
static EpiGPU * epi_h;
static EpjGPU * epj_d;
static EpjGPU * epj_h;
static int * ni_disp_d;
static int * ni_disp_h;
static int * nj_disp_d;
static int * nj_disp_h;

int DispatchKernelWithSP(const PS::S32 tag,
                         const int    n_walk,
                         const EPIGrav ** epi,
                         const int  *  n_epi,
                         const EPJGrav ** epj,
                         const int  *  n_epj,
                         const PS::SPJMonopole ** spj,
                         const int  *  n_spj){

    static bool first = true;
    assert(n_walk <= N_WALK_LIMIT);
    if(first){
        CUDA_SAFE_CALL( hipMalloc(     (void**)&ni_disp_d,  (N_WALK_LIMIT+1)*sizeof(int) ) );
        CUDA_SAFE_CALL( hipMalloc(     (void**)&nj_disp_d,  (N_WALK_LIMIT+1)*sizeof(int) ) );
        CUDA_SAFE_CALL( hipHostMalloc( (void**)&ni_disp_h,  (N_WALK_LIMIT+1)*sizeof(int) ) );
        CUDA_SAFE_CALL( hipHostMalloc( (void**)&nj_disp_h,  (N_WALK_LIMIT+1)*sizeof(int) ) );
        CUDA_SAFE_CALL( hipMalloc( (void**)&epi_d,       NI_LIMIT*sizeof(EpiGPU) ) );
        CUDA_SAFE_CALL( hipMalloc( (void**)&epj_d,       NJ_LIMIT*sizeof(EpjGPU) ) );
        CUDA_SAFE_CALL( hipMalloc( (void**)&force_d,     NI_LIMIT*sizeof(ForceGPU) ) );
        CUDA_SAFE_CALL( hipHostMalloc( (void**)&epi_h,   NI_LIMIT*sizeof(EpiGPU) ) );
        CUDA_SAFE_CALL( hipHostMalloc( (void**)&epj_h,   NJ_LIMIT*sizeof(EpjGPU) ) );
        CUDA_SAFE_CALL( hipHostMalloc( (void**)&force_h, NI_LIMIT*sizeof(ForceGPU) ) );
        first = false;
    }
    const float eps2 = EPIGrav::eps * EPIGrav::eps;
    //CUDA_SAFE_CALL( hipMalloc(     (void**)&ni_disp_d,     (n_walk+1)*sizeof(int) ) );
    //CUDA_SAFE_CALL( hipMalloc(     (void**)&nj_disp_d,     (n_walk+1)*sizeof(int) ) );
    //CUDA_SAFE_CALL( hipHostMalloc( (void**)&ni_disp_h,     (n_walk+1)*sizeof(int) ) );
    //CUDA_SAFE_CALL( hipHostMalloc( (void**)&nj_disp_h,     (n_walk+1)*sizeof(int) ) );

    ni_disp_h[0] = nj_disp_h[0] = 0;
    for(int i=0; i<n_walk; i++){
        ni_disp_h[i+1] = ni_disp_h[i] + n_epi[i];
        nj_disp_h[i+1] = nj_disp_h[i] + n_epj[i] + n_spj[i];
    }
    int ni_tot = ni_disp_h[n_walk];
    const int ni_tot_reg = ni_disp_h[n_walk] + ( (ni_tot%N_THREAD_GPU != 0) ? (N_THREAD_GPU - (ni_tot%N_THREAD_GPU)) : 0);
    //CUDA_SAFE_CALL( hipMalloc( (void**)&epi_d, ni_tot_reg*sizeof(EpiGPU) ) );
    //CUDA_SAFE_CALL( hipMalloc( (void**)&epj_d, nj_disp_h[n_walk]*sizeof(EpjGPU) ) );
    //CUDA_SAFE_CALL( hipMalloc( (void**)&force_d,     ni_tot_reg*sizeof(ForceGPU) ) );
    //CUDA_SAFE_CALL( hipHostMalloc( (void**)&epi_h, ni_tot_reg*sizeof(EpiGPU) ) );
    //CUDA_SAFE_CALL( hipHostMalloc( (void**)&epj_h, nj_disp_h[n_walk]*sizeof(EpjGPU) ) );
    //CUDA_SAFE_CALL( hipHostMalloc( (void**)&force_h, ni_tot_reg*sizeof(ForceGPU) ) );
    assert(ni_tot_reg <= NI_LIMIT);
    assert(nj_disp_h[n_walk] <= NJ_LIMIT);
    ni_tot = 0;
    int nj_tot = 0;
    for(int iw=0; iw<n_walk; iw++){
        for(int ip=0; ip<n_epi[iw]; ip++){
            epi_h[ni_tot].pos[0]  = float2_split(epi[iw][ip].pos.x);
            epi_h[ni_tot].pos[1]  = float2_split(epi[iw][ip].pos.y);
            epi_h[ni_tot].pos[2]  = float2_split(epi[iw][ip].pos.z);
            epi_h[ni_tot].id_walk = iw;
            force_h[ni_tot].acc[0] = force_h[ni_tot].acc[1] 
                = force_h[ni_tot].acc[2] = force_h[ni_tot].pot = make_float2(0.0, 0.0);
            ni_tot++;
        }
        for(int jp=0; jp<n_epj[iw]; jp++){
            epj_h[nj_tot].mass    = epj[iw][jp].mass;
            epj_h[nj_tot].pos[0]  = float2_split(epj[iw][jp].pos.x);
            epj_h[nj_tot].pos[1]  = float2_split(epj[iw][jp].pos.y);
            epj_h[nj_tot].pos[2]  = float2_split(epj[iw][jp].pos.z);
            nj_tot++;
        }
        for(int jp=0; jp<n_spj[iw]; jp++){
            epj_h[nj_tot].mass    = spj[iw][jp].getCharge();
            epj_h[nj_tot].pos[0]  = float2_split(spj[iw][jp].getPos().x);
            epj_h[nj_tot].pos[1]  = float2_split(spj[iw][jp].getPos().y);
            epj_h[nj_tot].pos[2]  = float2_split(spj[iw][jp].getPos().z);
            nj_tot++;
        }
    }
    for(int ip=ni_tot; ip<ni_tot_reg; ip++){
        epi_h[ni_tot].pos[0]  = epi_h[ni_tot].pos[1]  = epi_h[ni_tot].pos[2]  = make_float2(0.0, 0.0);
        epi_h[ni_tot].id_walk = 0;
        force_h[ni_tot].acc[0] = force_h[ni_tot].acc[1] 
            = force_h[ni_tot].acc[2] = force_h[ni_tot].pot = make_float2(0.0, 0.0);
    }
    CUDA_SAFE_CALL( hipMemcpy(epi_d, epi_h, ni_tot_reg*sizeof(EpiGPU), hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemcpy(epj_d, epj_h, nj_tot*sizeof(EpjGPU), hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemcpy(ni_disp_d, ni_disp_h, (n_walk+1)*sizeof(int), hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemcpy(nj_disp_d, nj_disp_h, (n_walk+1)*sizeof(int), hipMemcpyHostToDevice) );
    const int n_grid = ni_tot_reg/N_THREAD_GPU + ((ni_tot_reg%N_THREAD_GPU == 0) ? 0 : 1);
    dim3 size_grid(n_grid, 1, 1);
    dim3 size_thread(N_THREAD_GPU, 1, 1);
    ForceKernel<<<size_grid, size_thread>>> (epi_d, ni_disp_d, epj_d, nj_disp_d, force_d, float(eps2));

    return 0;
}

int RetrieveKernel(const PS::S32 tag,
                   const PS::S32    n_walk,
                   const PS::S32 *  ni,
                   ForceGrav     ** force){

    int ni_tot = 0;
    for(int i=0; i<n_walk; i++){
        ni_tot += ni[i];
    }
    //const int ni_tot_reg = ni_disp_h[n_walk] + ( (ni_tot%N_THREAD_GPU != 0) ? (N_THREAD_GPU - (ni_tot%N_THREAD_GPU)) : 0);
    //CUDA_SAFE_CALL( hipMemcpy(force_h, force_d,      ni_tot_reg*sizeof(ForceGPU), hipMemcpyDeviceToHost) );
    CUDA_SAFE_CALL( hipMemcpy(force_h, force_d,      ni_tot*sizeof(ForceGPU), hipMemcpyDeviceToHost) );
    int n_cnt = 0;
    for(int iw=0; iw<n_walk; iw++){
        for(int ip=0; ip<ni[iw]; ip++){
            force[iw][ip].acc.x = (double)force_h[n_cnt].acc[0].x + (double)force_h[n_cnt].acc[0].y;
            force[iw][ip].acc.y = (double)force_h[n_cnt].acc[1].x + (double)force_h[n_cnt].acc[1].y;
            force[iw][ip].acc.z = (double)force_h[n_cnt].acc[2].x + (double)force_h[n_cnt].acc[2].y;
            force[iw][ip].pot   = (double)force_h[n_cnt].pot.x    + (double)force_h[n_cnt].pot.y;
            force[iw][ip].pot *= -1.0;
            n_cnt++;
        }
    }

    /*
    CUDA_SAFE_CALL( hipHostFree(force_h) );
    CUDA_SAFE_CALL( hipFree(force_d) );
    CUDA_SAFE_CALL( hipFree(epi_d) );
    CUDA_SAFE_CALL( hipHostFree(epi_h) );
    CUDA_SAFE_CALL( hipFree(epj_d) );
    CUDA_SAFE_CALL( hipHostFree(epj_h) );
    CUDA_SAFE_CALL( hipFree(ni_disp_d) );
    CUDA_SAFE_CALL( hipHostFree(ni_disp_h) );
    CUDA_SAFE_CALL( hipFree(nj_disp_d) );
    CUDA_SAFE_CALL( hipHostFree(nj_disp_h) );
    */
    return 0;
}
